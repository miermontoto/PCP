#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

__global__ void kernelScal(double *A, double alpha, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    A[i] = A[i]*alpha;
}


extern "C" int ScalGPU(double *x, const double alpha, const int n, const int ThpBlk) 
{
   int    NumGPUs, NumBlk;
   double *device_x=NULL;

   if(hipSuccess != hipGetDeviceCount(&NumGPUs))
     return -1;
   
   hipMalloc((void **)&device_x, sizeof(double)*n);
   hipMemcpy(device_x, x, sizeof(double)*n, hipMemcpyHostToDevice);

   NumBlk=(n + ThpBlk - 1) / ThpBlk;

   kernelScal<<<NumBlk, ThpBlk>>>(device_x, alpha, n);

   hipMemcpy(x, device_x, sizeof(double)*n, hipMemcpyDeviceToHost);
     
   return 0;
}
