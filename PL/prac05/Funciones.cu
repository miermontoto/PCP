#include "hip/hip_runtime.h"
#include "Prototipos.h"

double Ctimer(void)
{
  struct timeval tm;

  gettimeofday(&tm, NULL);

  return tm.tv_sec + tm.tv_usec/1.0E6;
}



/* Rellena por filas los elementos de una matriz A de dimensiones m x n */
void Genera(double *A, int n, int seed) {
   int i=0;

   srand(seed);

   for (i=0; i<n; i++)
     A[i] = ((double)(rand()% 1000 + 1))/1.0E3;
}

/* Calculando el error con norma Frobenius */
double Error(int n, double *X, double *y)
{
   int i;

   double tmp, error=0.0;

   for (i=0; i<n; i++)
   {
      tmp = X[i] - y[i];
      error += tmp*tmp;
   }

   return sqrt(error/n);
}


__global__ void kernel_VecAdd(double *v, const double *x, const double *y, const int size)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   if (tid < size)
      v[tid] = x[tid] + y[tid];
}


__global__ void kernel_VecAdd1D(double *v, const double *x, const double *y, const int size)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   if (tid < size)
      v[tid] = x[tid] + y[tid];
}


__global__ void kernel_VecAdd2D(double *v, const double *x, const double *y, const int rows, const int cols)
{
   int X = blockIdx.x * blockDim.x + threadIdx.x;
   int Y = blockIdx.y * blockDim.y + threadIdx.y;

   if (Y<rows && X<cols)
      v[Y*cols + X] = x[Y*cols + X] + y[Y*cols + X];
}

__global__ void kernel6_1(double *x, double *y, const int size) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   if (tid < size) {
      x[tid] = y[tid] * y[tid] + x[tid];
   }
}

__global__ void kernel6_2(double *x, double *y, double *A, const int n) {

   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   if (tid < n) {
      y[tid] = 0.0;
      for (int i=0; i<n; i++)
         y[tid] += A[tid*n + i] * x[i];
   }
}
