#include "Prototipos.h"
#include "Kernels.h"

int main(int argc, char *argv[]) {

    double *Hx, *Hy, *Hs;
    int ndev;
    double *Dx, *Dy, *Dv;
    float milliseconds;

    hipEvent_t start, stop;

    int n = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int veces = atoi(argv[3]);
    int seed = atoi(argv[4]);

    CHECKNULL(Hx = (double *)malloc(n * sizeof(double)));
    CHECKNULL(Hy = (double *)malloc(n * sizeof(double)));
    CHECKNULL(Hs = (double *)malloc(n * sizeof(double)));

    Genera(Hx, n, seed);
    Genera(Hy, n, seed+18);

    hipError_t error = hipGetDeviceCount(&ndev);
    if (ndev == 0 || error != 0) {
        printf("No hay dispositivos CUDA disponibles");
        return EXIT_FAILURE;
    }

    CUDAERR(hipMalloc((void **)&Dx, n * sizeof(double)));
    CUDAERR(hipMalloc((void **)&Dy, n * sizeof(double)));

    CUDAERR(hipMemcpy(Dx, Hx, n * sizeof(double), hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(Dy, Hy, n * sizeof(double), hipMemcpyHostToDevice));

    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // cpu
    for (int i = 0; i < veces; i++) {
        for (int j = 0; j < n; j++) {
            Hx[j] = Hx[j] + Hy[j];
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CPU: %f ms\n", milliseconds);

    hipEventRecord(start);
    for (int i = 0; i < veces; i++) {
        kernel6_1<<<numBlocks, threadsPerBlock>>>(Dx, Dy, n);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA: %f ms\n", milliseconds);

    CUDAERR(hipMemcpy(Hs, Dx, n * sizeof(double), hipMemcpyDeviceToHost));
    printf("\nError: %2.7E\n", Error(n, Hs, Hx));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(Dx);
    hipFree(Dy);
    free(Hx);
    free(Hy);
    free(Hs);
}