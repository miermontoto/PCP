#include "Prototipos.h"

int main(int argc, char *argv[]) {

    double *Hx, *Hy, *Hs;
    int ndev;
    float milliseconds;

    hipEvent_t start, stop;

    int n = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int veces = atoi(argv[3]);
    int seed = atoi(argv[4]);

    CHECKNULL(Hx = (double *)malloc(n * sizeof(double)));
    CHECKNULL(Hy = (double *)malloc(n * sizeof(double)));
    CHECKNULL(Hs = (double *)malloc(n * sizeof(double)));

    Genera(Hx, n, seed);
    Genera(Hy, n, seed+18);

    hipError_t error = hipGetDeviceCount(&ndev);
    if (ndev == 0 || error != 0) {
        printf("No hay dispositivos CUDA disponibles");
        return EXIT_FAILURE;
    }

    CUDAERR(hipMallocManaged((void**)&Hx, n * sizeof(double)));
    CUDAERR(hipMallocManaged((void**)&Hy, n * sizeof(double)));
    CUDAERR(hipMallocManaged((void**)&Hs, n * sizeof(double)));

    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // cpu
    for (int i = 0; i < veces; i++) {
        for (int j = 0; j < n; j++) {
            Hs[j] = Hx[j] + Hy[j];
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CPU: %f ms\n", milliseconds);

    hipEventRecord(start);
    for (int i = 0; i < veces; i++) {
        kernel6_1<<<numBlocks, threadsPerBlock>>>(Hx, Hy, n);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA: %f ms\n", milliseconds);

    printf("\nError: %2.7E\n", Error(n, Hs, Hx));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(Hx);
    hipFree(Hy);
    hipFree(Hs);
}