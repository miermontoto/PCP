#include "Prototipos.h"

int main(int argc, char *argv[]) {

    double *Hx, *Hy, *HA, *Hs;
    int ndev;
    double *Dx, *Dy, *DA;
    float milliseconds;

    hipEvent_t start, stop;

    int n = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int veces = atoi(argv[3]);
    int seed = atoi(argv[4]);

    CHECKNULL(Hx = (double *)malloc(n * sizeof(double)));
    CHECKNULL(Hy = (double *)malloc(n * sizeof(double)));
    CHECKNULL(Hs = (double *)malloc(n * sizeof(double)));

    CHECKNULL(HA = (double *)malloc(n * n * sizeof(double)));

    Genera(Hx, n, seed);
    Genera(Hy, n, seed+18);
    Genera(HA, n*n, seed+36);

    hipError_t error = hipGetDeviceCount(&ndev);
    if (ndev == 0 || error != 0) {
        printf("No hay dispositivos CUDA disponibles");
        return EXIT_FAILURE;
    }

    CUDAERR(hipMalloc((void **)&Dx, n * sizeof(double)));
    CUDAERR(hipMalloc((void **)&Dy, n * sizeof(double)));
    CUDAERR(hipMalloc((void **)&DA, n * n * sizeof(double)));

    CUDAERR(hipMemcpy(Dx, Hx, n * sizeof(double), hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(Dy, Hy, n * sizeof(double), hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(DA, HA, n * n * sizeof(double), hipMemcpyHostToDevice));

    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // cpu
    for (int i = 0; i < veces; i++) {
        for (int j = 0; j < n; j++) {
            Hs[j] = 0;
            for (int k = 0; k < n; k++) {
                Hs[j] += HA[j * n + k] * Hx[k];
            }
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CPU: %f ms\n", milliseconds);

    hipEventRecord(start);
    for (int i = 0; i < veces; i++) {
        kernel6_2<<<numBlocks, threadsPerBlock>>>(Dx, Dy, DA, n);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA: %f ms\n", milliseconds);

    CUDAERR(hipMemcpy(Hx, Dy, n * sizeof(double), hipMemcpyDeviceToHost));
    printf("\nError: %2.7E\n", Error(n, Hs, Hx));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(Dx);
    hipFree(Dy);
    hipFree(DA);
    free(Hx);
    free(Hy);
    free(HA);
    free(Hs);
}