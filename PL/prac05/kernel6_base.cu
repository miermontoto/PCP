#include "Prototipos.h"
#include "Kernels.h"

void kernel6_1(int, int, int, hipEvent_t, hipEvent_t, bool);
void kernel6_2(int, int, int, hipEvent_t, hipEvent_t);

int main(int argc, char *argv[]) {

    int ndev;
    float milliseconds;
    hipEvent_t start, stop;

    int n = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int repetitions = atoi(argv[3]);

    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipError_t rev = hipSetDevice(hipGetDeviceCount(&ndev));
    if (ndev == 0 || rev != hipSuccess) {
        printf("No CUDA devices found");
        return EXIT_FAILURE;
    }

    kernel6_1(n, threadsPerBlock, repetitions, start, stop, false);


}

void kernel6_1(int size, int tpb, int rept, hipEvent_t start, hipEvent_t stop, bool shared) {
    double *Hx, *Hy, *Hs;
    double *Dx, *Dy, *Dv;

    unsigned int seed;
    srand(seed);

    CHECKNULL(Hx = (double *)malloc(size * sizeof(double)));
    CHECKNULL(Hy = (double *)malloc(size * sizeof(double)));
    CHECKNULL(Hs = (double *)malloc(size * sizeof(double)));

    Genera(Hx, size, seed);
    Genera(Hy, size, seed+18);

    CUDAERR(hipMalloc((void **)&Dx, size * sizeof(double)));
    CUDAERR(hipMalloc((void **)&Dy, size * sizeof(double)));

    CUDAERR(hipMemcpy(Dx, Hx, size * sizeof(double), hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(Dy, Hy, size * sizeof(double), hipMemcpyHostToDevice));

    int numBlocks = (size + tpb - 1) / tpb;

    // CPU
    for(int i = 0; i < rept; i++) {
        hipEventRecord(start);
        for(int j = 0; j < size; j++) {
            Hs[j] = Hx[j] + Hy[j];
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("CPU: %f ms", milliseconds);
    }

    // CUDA
    for(int i = 0; i < rept; i++) {
        hipEventRecord(start);
        if(shared) {
            kernel6_1Sh<<<numBlocks, tpb>>>(size, Dx, Dy, Dv);
        } else {
            kernel6_1<<<numBlocks, tpb>>>(size, Dx, Dy, Dv);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("CUDA: %f ms", milliseconds);
    }

}

