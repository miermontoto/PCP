#include "Prototipos.h"

int main(int argc, char *argv[])
{
  int
    n, seed, veces, i, j;
  
  double
    *Host_x = NULL,
    *Host_y = NULL,
    *Host_v = NULL,
    *Host_s = NULL;

  /* CUDA and CUBLAS variables */
  int
    ndev,
    ThPerBlk,
    numBlocks;

  /* BEGIN NEW */    
  //double
  //  *Devi_x = NULL,
  //  *Devi_y = NULL,
  //  *Devi_v = NULL;
  /* END NEW*/
  
  hipEvent_t 
    start, stop;
    
  float
    time;
  
  if (argc != 5) {
     printf("Uso: %s <n> <hilos por bloque> <veces> <seed>\n", argv[0]);
     return -1;
  }

  n        = atoi(argv[1]);
  ThPerBlk = atoi(argv[2]);
  veces    = atoi(argv[3]);
  seed     = atoi(argv[4]);

  hipError_t ret=hipGetDeviceCount(&ndev);
  if (ndev == 0||ret!=0)
  {
     printf("Error 1: No hay GPU con capacidades CUDA\n");
     return -1;
  }else
     printf("INFO: Hay %d GPUs con capacidades CUDA, seguimos\n", ndev);  

  /* BEGIN NEW */
  CUDAERR(hipMallocManaged((void**)&Host_x, n*sizeof(double), hipMemAttachGlobal));
  CUDAERR(hipMallocManaged((void**)&Host_y, n*sizeof(double), hipMemAttachGlobal));
  CUDAERR(hipMallocManaged((void**)&Host_v, n*sizeof(double), hipMemAttachGlobal));
  CUDAERR(hipMallocManaged((void**)&Host_s, n*sizeof(double), hipMemAttachGlobal));
  /* END NEW */

  Genera(Host_x, n, seed);
  Genera(Host_y, n, seed+11);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Resuelve el problema en la CPU */
  hipEventRecord(start, 0);
     for (i=1; i<=veces; i++)
        for (j=0; j<n; j++) Host_s[j] = Host_x[j] + Host_y[j]; /* OJO, el resultado es Host_s */
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("El tiempo en la CPU     es %2.7E segundos.\n", time/1000.0);

  /* Resuelve el problema en la GPU */
  numBlocks = (n + ThPerBlk - 1) / ThPerBlk;
  hipEventRecord(start, 0);
     for (i=1; i<=veces; i++)
        kernel_VecAdd<<<numBlocks, ThPerBlk>>>(Host_v, Host_x, Host_y, n);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  CHECKLASTERR();
  printf("El tiempo en la GPU     es %2.7E segundos.\n", time/1000.0);

  printf("El error es %2.7E.\n", Error(n, Host_s, Host_v));

  /* BEGIN NEW */
  CUDAERR(hipFree(Host_x));
  CUDAERR(hipFree(Host_y));
  CUDAERR(hipFree(Host_v));
  CUDAERR(hipFree(Host_s));
  /* END NEW */

  return 0;
}
